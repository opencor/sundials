#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles, and Cody J. Balos @ LLNL
 * -----------------------------------------------------------------
 * SUNDIALS Copyright Start
 * Copyright (c) 2002-2019, Lawrence Livermore National Security
 * and Southern Methodist University.
 * All rights reserved.
 *
 * See the top-level LICENSE and NOTICE files for details.
 *
 * SPDX-License-Identifier: BSD-3-Clause
 * SUNDIALS Copyright End
 * -----------------------------------------------------------------
 * This is the testing routine to check the NVECTOR CUDA module
 * implementation.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>

#include <sundials/sundials_types.h>
#include <nvector/cuda/Vector.hpp>
#include <nvector/nvector_mpicuda.h>
#include <sundials/sundials_math.h>
#include "test_nvector.h"

#include <mpi.h>

/* CUDA vector specific tests */
int Test_N_VMake_Cuda(N_Vector X, sunindextype local_length, sunindextype global_length, MPI_Comm comm, int myid);
int Test_N_VMakeManaged_Cuda(N_Vector X, sunindextype local_length, sunindextype global_length, MPI_Comm comm, int myid);

/* CUDA vector can use unmanaged or managed memory */
enum mem_type { UNMANAGED, MANAGED };

/* ----------------------------------------------------------------------
 * Main NVector Testing Routine
 * --------------------------------------------------------------------*/
int main(int argc, char *argv[])
{
  int          fails = 0;         /* counter for test failures */
  int          globfails = 0;     /* counter for test failures */
  int          retval;            /* function return value     */
  sunindextype local_length;      /* local vector length       */
  sunindextype global_length;     /* global vector length      */
  N_Vector     U, V, X, Y, Z;     /* test vectors              */
  int          print_timing;      /* turn timing on/off        */
  MPI_Comm     comm;              /* MPI Communicator          */
  int          nprocs, myid;      /* Number of procs, proc id  */
  int          i;

  /* Get processor number and total number of processes */
  MPI_Init(&argc, &argv);
  comm = MPI_COMM_WORLD;
  MPI_Comm_size(comm, &nprocs);
  MPI_Comm_rank(comm, &myid);

  /* check inputs */
  if (argc < 3) {
    if (myid == 0)
      printf("ERROR: TWO (2) Inputs required: vector length, print timing \n");
    MPI_Abort(comm, -1);
  }

  local_length = atol(argv[1]);
  if (local_length < 1) {
    if (myid == 0)
      printf("ERROR: local vector length must be a positive integer \n");
    MPI_Abort(comm, -1);
  }

  print_timing = atoi(argv[2]);
  SetTiming(print_timing, myid);

  /* global length */
  global_length = nprocs*local_length;

  for (i=UNMANAGED; i<=MANAGED; ++i) {
    if (myid == 0) {
      if (i==UNMANAGED) {
        printf("Testing CUDA N_Vector \n");
      } else {
        printf("\nTesting CUDA N_Vector with managed memory \n");
      }
      printf("Vector global length %ld \n", (long int) global_length);
      printf("MPI processes %d \n", nprocs);
    }

    /* Create new vectors */
    X = (i==UNMANAGED) ? N_VNew_Cuda(comm, local_length, global_length) : 
                         N_VNewManaged_Cuda(comm, local_length, global_length);
    if (X == NULL) {
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* Check vector ID */
    fails += Test_N_VGetVectorID(X, SUNDIALS_NVEC_CUDA, myid);

    /* Test clone functions */
    fails += Test_N_VCloneEmpty(X, myid);
    fails += Test_N_VClone(X, local_length, myid);
    fails += Test_N_VCloneEmptyVectorArray(5, X, myid);
    fails += Test_N_VCloneVectorArray(5, X, local_length, myid);

    /* Clone additional vectors for testing */
    Y = N_VClone(X);
    if (Y == NULL) {
      N_VDestroy(X);
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    Z = N_VClone(X);
    if (Z == NULL) {
      N_VDestroy(X);
      N_VDestroy(Y);
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* Standard vector operation tests */
    if (myid == 0) printf("\nTesting standard vector operations:\n\n");

    fails += Test_N_VConst(X, local_length, myid);
    fails += Test_N_VLinearSum(X, Y, Z, local_length, myid);
    fails += Test_N_VProd(X, Y, Z, local_length, myid);
    fails += Test_N_VDiv(X, Y, Z, local_length, myid);
    fails += Test_N_VScale(X, Z, local_length, myid);
    fails += Test_N_VAbs(X, Z, local_length, myid);
    fails += Test_N_VInv(X, Z, local_length, myid);
    fails += Test_N_VAddConst(X, Z, local_length, myid);
    fails += Test_N_VDotProd(X, Y, local_length, global_length, myid);
    fails += Test_N_VMaxNorm(X, local_length, myid);
    fails += Test_N_VWrmsNorm(X, Y, local_length, myid);
    fails += Test_N_VWrmsNormMask(X, Y, Z, local_length, global_length, myid);
    fails += Test_N_VMin(X, local_length, myid);
    fails += Test_N_VWL2Norm(X, Y, local_length, global_length, myid);
    fails += Test_N_VL1Norm(X, local_length, global_length, myid);
    fails += Test_N_VCompare(X, Z, local_length, myid);
    fails += Test_N_VInvTest(X, Z, local_length, myid);
    fails += Test_N_VConstrMask(X, Y, Z, local_length, myid);
    fails += Test_N_VMinQuotient(X, Y, local_length, myid);

    /* Fused and vector array operations tests (disabled) */
    if (myid == 0) printf("\nTesting fused and vector array operations (disabled):\n\n");

    /* create vector and disable all fused and vector array operations */
    U = (i==UNMANAGED) ? N_VNew_Cuda(comm, local_length, global_length) : 
                         N_VNewManaged_Cuda(comm, local_length, global_length);
    retval = N_VEnableFusedOps_Cuda(U, SUNFALSE);
    if (U == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(U, local_length, myid);
    fails += Test_N_VScaleAddMulti(U, local_length, myid);
    fails += Test_N_VDotProdMulti(U, local_length, global_length, myid);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(U, local_length, myid);
    fails += Test_N_VScaleVectorArray(U, local_length, myid);
    fails += Test_N_VConstVectorArray(U, local_length, myid);
    fails += Test_N_VWrmsNormVectorArray(U, local_length, myid);
    fails += Test_N_VWrmsNormMaskVectorArray(U, local_length, global_length, myid);
    fails += Test_N_VScaleAddMultiVectorArray(U, local_length, myid);
    fails += Test_N_VLinearCombinationVectorArray(U, local_length, myid);

    /* Fused and vector array operations tests (enabled) */
    if (myid == 0) printf("\nTesting fused and vector array operations (enabled):\n\n");

    /* create vector and enable all fused and vector array operations */
    V = (i==UNMANAGED) ? N_VNew_Cuda(comm, local_length, global_length) : 
                         N_VNewManaged_Cuda(comm, local_length, global_length);
    retval = N_VEnableFusedOps_Cuda(V, SUNTRUE);
    if (V == NULL || retval != 0) {
      N_VDestroy(X);
      N_VDestroy(Y);
      N_VDestroy(Z);
      N_VDestroy(U);
      if (myid == 0) printf("FAIL: Unable to create a new vector \n\n");
      MPI_Abort(comm, 1);
    }

    /* fused operations */
    fails += Test_N_VLinearCombination(V, local_length, myid);
    fails += Test_N_VScaleAddMulti(V, local_length, myid);
    fails += Test_N_VDotProdMulti(V, local_length, global_length, myid);

    /* vector array operations */
    fails += Test_N_VLinearSumVectorArray(V, local_length, myid);
    fails += Test_N_VScaleVectorArray(V, local_length, myid);
    fails += Test_N_VConstVectorArray(V, local_length, myid);
    fails += Test_N_VWrmsNormVectorArray(V, local_length, myid);
    fails += Test_N_VWrmsNormMaskVectorArray(V, local_length, global_length, myid);
    fails += Test_N_VScaleAddMultiVectorArray(V, local_length, myid);
    fails += Test_N_VLinearCombinationVectorArray(V, local_length, myid);

    /* CUDA specific tests */
    if (i==UNMANAGED) {
      fails += Test_N_VMake_Cuda(X, local_length, global_length, comm, myid);
    } else {
      fails += Test_N_VMakeManaged_Cuda(X, local_length, global_length, comm, myid);
    }

    /* Free vectors */
    N_VDestroy(X);
    N_VDestroy(Y);
    N_VDestroy(Z);
    N_VDestroy(U);
    N_VDestroy(V);
  }

  /* Print result */
  if (fails) {
    printf("FAIL: NVector module failed %i tests, Proc %d \n\n", fails, myid);
  } else {
    if (myid == 0) 
      printf("SUCCESS: NVector module passed all tests \n\n");
  }

  /* check if any other process failed */
  (void) MPI_Allreduce(&fails, &globfails, 1, MPI_INT, MPI_MAX, comm);
  
  MPI_Finalize();

  return(globfails);
}


/* ----------------------------------------------------------------------
 * CUDA specific tests
 * --------------------------------------------------------------------*/

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMake_Cuda function. Requires N_VConst
 * to check data.
 */
int Test_N_VMake_Cuda(N_Vector X, sunindextype local_length, sunindextype global_length, MPI_Comm comm, int myid)
{
  int failure = 0;
  realtype *h_data, *d_data;
  N_Vector Y;

  N_VConst(NEG_HALF, X);
  N_VCopyFromDevice_Cuda(X);

  h_data = N_VGetHostArrayPointer_Cuda(X);
  d_data = N_VGetDeviceArrayPointer_Cuda(X);

  /* Case 1: h_data and d_data are not null */
  Y = N_VMake_Cuda(comm, local_length, global_length, h_data, d_data);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  if (N_VGetHostArrayPointer_Cuda(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector host data == NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }
  
  if (N_VGetDeviceArrayPointer_Cuda(Y) == NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda, Proc %d \n", myid);
    printf("    Vector device data -= NULL \n \n");
    N_VDestroy(Y);
    return(1);
  }
  
  failure += check_ans(NEG_HALF, Y, local_length);
 
  if (failure) {
    printf(">>> FAILED test -- N_VMake_Cuda Case 1, Proc %d \n", myid);
    printf("    Failed N_VConst check \n \n");
    N_VDestroy(Y);
    return(1);
  }
  
  if (myid == 0) {
    printf("PASSED test -- N_VMake_Cuda Case 1 \n");
  }

  N_VDestroy(Y);

  /* Case 2: data is null */
  Y = N_VMake_Cuda(comm, local_length, global_length, NULL, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMake_Cuda Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMake_Cuda Case 2 \n");
  }
  
  N_VDestroy(Y);

  return(failure);
}

/* --------------------------------------------------------------------
 * Test for the CUDA N_Vector N_VMakeManaged_Cuda function. Requires
 * N_VConst to check data. X must be using managed memory.
 */
int Test_N_VMakeManaged_Cuda(N_Vector X, sunindextype local_length, sunindextype global_length, MPI_Comm comm, int myid)
{
  int failure = 0;
  realtype *vdata;
  N_Vector Y;

  if(!N_VIsManagedMemory_Cuda(X)) {
    printf(">>> FAILED test -- N_VIsManagedMemory_Cuda, Proc %d \n", myid);
    return(1);
  }

  N_VConst(NEG_HALF, X);

  vdata = N_VGetHostArrayPointer_Cuda(X);
  
  /* Case 1: data is not null */
  Y = N_VMakeManaged_Cuda(comm, local_length, global_length, vdata);
  if (Y == NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda, Proc %d \n", myid);
    printf("    Vector is NULL \n \n");
    return(1);
  }

  failure += check_ans(NEG_HALF, Y, local_length);
 
  /* Case 2: data is null */
  Y = N_VMakeManaged_Cuda(comm, local_length, global_length, NULL);
  if (Y != NULL) {
    printf(">>> FAILED test -- N_VMakeManaged_Cuda Case 2, Proc %d \n", myid);
    printf("    Vector is not NULL \n \n");
    return(1);
  }

  if (myid == 0) {
    printf("PASSED test -- N_VMakeManaged_Cuda Case 2 \n");
  }
  
  N_VDestroy(Y);
 
  return(failure);
}


/* ----------------------------------------------------------------------
 * Implementation specific utility functions for vector tests
 * --------------------------------------------------------------------*/
int check_ans(realtype ans, N_Vector X, sunindextype local_length)
{
  int          failure = 0;
  sunindextype i;
  realtype     *Xdata;

  N_VCopyFromDevice_Cuda(X);
  Xdata = N_VGetHostArrayPointer_Cuda(X);

  /* check vector data */
  for (i = 0; i < local_length; i++) {
    failure += FNEQ(Xdata[i], ans);
  }

  return (failure > ZERO) ? (1) : (0);
}

booleantype has_data(N_Vector X)
{
  /* check if vector content is non-null */
  return (X->content == NULL ? SUNFALSE : SUNTRUE);
}

void set_element(N_Vector X, sunindextype i, realtype val)
{
  /* set i-th element of data array */
  N_VCopyFromDevice_Cuda(X);
  (N_VGetHostArrayPointer_Cuda(X))[i] = val;
  N_VCopyToDevice_Cuda(X);
}

realtype get_element(N_Vector X, sunindextype i)
{
  /* get i-th element of data array */
  N_VCopyFromDevice_Cuda(X);
  return (N_VGetHostArrayPointer_Cuda(X))[i];
}

double max_time(N_Vector X, double time)
{
  MPI_Comm comm;
  double maxt;
 
  comm = N_VGetMPIComm_Cuda(X);
  
  /* get max time across all MPI ranks */
  (void) MPI_Reduce(&time, &maxt, 1, MPI_DOUBLE, MPI_MAX, 0, comm);
  return(maxt);
}

void sync_device()
{
  /* sync with GPU */
  hipDeviceSynchronize();
  return;
}
