#include "hip/hip_runtime.h"
/* -----------------------------------------------------------------
 * Programmer(s): Slaven Peles @ LLNL
 * -----------------------------------------------------------------
 * LLNS Copyright Start
 * Copyright (c) 2014, Lawrence Livermore National Security
 * This work was performed under the auspices of the U.S. Department
 * of Energy by Lawrence Livermore National Laboratory in part under
 * Contract W-7405-Eng-48 and in part under Contract DE-AC52-07NA27344.
 * Produced at the Lawrence Livermore National Laboratory.
 * All rights reserved.
 * For details, see the LICENSE file.
 * LLNS Copyright End
 * -----------------------------------------------------------------
 * This is the implementation file for a MPI+CUDA implementation
 * of the NVECTOR package.
 * -----------------------------------------------------------------*/

#include <stdio.h>
#include <stdlib.h>
#include <cmath>

#include <nvector/cuda/Vector.hpp>
#include <nvector/cuda/VectorKernels.cuh>
#include <sundials/sundials_mpi.h>

#define ZERO   RCONST(0.0)
#define HALF   RCONST(0.5)
#define ONE    RCONST(1.0)
#define ONEPT5 RCONST(1.5)

extern "C" {

using namespace suncudavec;

/*
 * Type definitions
 */

typedef suncudavec::Vector<realtype, sunindextype> vector_type;

/* ----------------------------------------------------------------
 * Returns vector type ID. Used to identify vector implementation
 * from abstract N_Vector interface.
 */
N_Vector_ID N_VGetVectorID_Cuda(N_Vector v)
{
  return SUNDIALS_NVEC_CUDA;
}

N_Vector N_VNewEmpty_Cuda(sunindextype length)
{
  N_Vector v;
  N_Vector_Ops ops;

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = N_VGetVectorID_Cuda;
  ops->nvclone           = N_VClone_Cuda;
  ops->nvcloneempty      = N_VCloneEmpty_Cuda;
  ops->nvdestroy         = N_VDestroy_Cuda;
  ops->nvspace           = N_VSpace_Cuda;
  ops->nvgetarraypointer = NULL;
  ops->nvsetarraypointer = NULL;
  ops->nvlinearsum       = N_VLinearSum_Cuda;
  ops->nvconst           = N_VConst_Cuda;
  ops->nvprod            = N_VProd_Cuda;
  ops->nvdiv             = N_VDiv_Cuda;
  ops->nvscale           = N_VScale_Cuda;
  ops->nvabs             = N_VAbs_Cuda;
  ops->nvinv             = N_VInv_Cuda;
  ops->nvaddconst        = N_VAddConst_Cuda;
  ops->nvdotprod         = N_VDotProd_Cuda;
  ops->nvmaxnorm         = N_VMaxNorm_Cuda;
  ops->nvwrmsnormmask    = N_VWrmsNormMask_Cuda;
  ops->nvwrmsnorm        = N_VWrmsNorm_Cuda;
  ops->nvmin             = N_VMin_Cuda;
  ops->nvwl2norm         = N_VWL2Norm_Cuda;
  ops->nvl1norm          = N_VL1Norm_Cuda;
  ops->nvcompare         = N_VCompare_Cuda;
  ops->nvinvtest         = N_VInvTest_Cuda;
  ops->nvconstrmask      = N_VConstrMask_Cuda;
  ops->nvminquotient     = N_VMinQuotient_Cuda;

  /* Attach ops and set content to NULL */
  v->content = NULL;
  v->ops     = ops;

  return(v);
}

#if SUNDIALS_MPI_ENABLED
N_Vector N_VNew_Cuda(MPI_Comm comm,
                     sunindextype local_length,
                     sunindextype global_length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda(local_length);
  if (v == NULL)
    return(NULL);

  v->content = new vector_type(comm, local_length, global_length);

  return(v);
}
#else
N_Vector N_VNew_Cuda(sunindextype length)
{
  N_Vector v;

  v = NULL;
  v = N_VNewEmpty_Cuda(length);
  if (v == NULL)
    return(NULL);

  v->content = new vector_type(SUNMPI_COMM_WORLD, length, length);

  return(v);
}
#endif


N_Vector N_VMake_Cuda(N_VectorContent_Cuda c)
{
  N_Vector v;
  vector_type* x = static_cast<vector_type*>(c);
  sunindextype length = x->size();

  v = NULL;
  v = N_VNewEmpty_Cuda(length);
  if (v == NULL) return(NULL);

  v->content = c;

  return(v);
}

/* -----------------------------------------------------------------
 * Function to return the length of the vector.
 */
sunindextype N_VGetLength_Cuda(N_Vector v)
{
  vector_type* xd = static_cast<vector_type*>(v->content);
  return xd->size();
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw host data
 */

realtype *N_VGetHostArrayPointer_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->host());
}

/* ----------------------------------------------------------------------------
 * Return pointer to the raw device data
 */

realtype *N_VGetDeviceArrayPointer_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  return (xv->device());
}

/* ----------------------------------------------------------------------------
 * Copy vector data to the device
 */

void N_VCopyToDevice_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyToDev();
}

/* ----------------------------------------------------------------------------
 * Copy vector data from the device to the host
 */

void N_VCopyFromDevice_Cuda(N_Vector x)
{
  vector_type* xv = static_cast<vector_type*>(x->content);
  xv->copyFromDev();
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to stdout
 */

void N_VPrint_Cuda(N_Vector x)
{
  N_VPrintFile_Cuda(x, stdout);
}

/* ----------------------------------------------------------------------------
 * Function to print the a CUDA-based vector to outfile
 */

void N_VPrintFile_Cuda(N_Vector x, FILE *outfile)
{
  sunindextype i;
  vector_type* xd = static_cast<vector_type*>(x->content);

  for (i = 0; i < xd->size(); i++) {
#if defined(SUNDIALS_EXTENDED_PRECISION)
    fprintf(outfile, "%35.32Lg\n", xd->host()[i]);
#elif defined(SUNDIALS_DOUBLE_PRECISION)
    fprintf(outfile, "%19.16g\n", xd->host()[i]);
#else
    fprintf(outfile, "%11.8g\n", xd->host()[i]);
#endif
  }
  fprintf(outfile, "\n");

  return;
}


/*
 * -----------------------------------------------------------------
 * implementation of vector operations
 * -----------------------------------------------------------------
 */

N_Vector N_VCloneEmpty_Cuda(N_Vector w)
{
  N_Vector v;
  N_Vector_Ops ops;

  if (w == NULL) return(NULL);

  /* Create vector */
  v = NULL;
  v = (N_Vector) malloc(sizeof *v);
  if (v == NULL) return(NULL);

  /* Create vector operation structure */
  ops = NULL;
  ops = (N_Vector_Ops) malloc(sizeof(struct _generic_N_Vector_Ops));
  if (ops == NULL) { free(v); return(NULL); }

  ops->nvgetvectorid     = w->ops->nvgetvectorid;
  ops->nvclone           = w->ops->nvclone;
  ops->nvcloneempty      = w->ops->nvcloneempty;
  ops->nvdestroy         = w->ops->nvdestroy;
  ops->nvspace           = w->ops->nvspace;
  ops->nvgetarraypointer = w->ops->nvgetarraypointer;
  ops->nvsetarraypointer = w->ops->nvsetarraypointer;
  ops->nvlinearsum       = w->ops->nvlinearsum;
  ops->nvconst           = w->ops->nvconst;
  ops->nvprod            = w->ops->nvprod;
  ops->nvdiv             = w->ops->nvdiv;
  ops->nvscale           = w->ops->nvscale;
  ops->nvabs             = w->ops->nvabs;
  ops->nvinv             = w->ops->nvinv;
  ops->nvaddconst        = w->ops->nvaddconst;
  ops->nvdotprod         = w->ops->nvdotprod;
  ops->nvmaxnorm         = w->ops->nvmaxnorm;
  ops->nvwrmsnormmask    = w->ops->nvwrmsnormmask;
  ops->nvwrmsnorm        = w->ops->nvwrmsnorm;
  ops->nvmin             = w->ops->nvmin;
  ops->nvwl2norm         = w->ops->nvwl2norm;
  ops->nvl1norm          = w->ops->nvl1norm;
  ops->nvcompare         = w->ops->nvcompare;
  ops->nvinvtest         = w->ops->nvinvtest;
  ops->nvconstrmask      = w->ops->nvconstrmask;
  ops->nvminquotient     = w->ops->nvminquotient;

  /* Create content */
  v->content = NULL;
  v->ops  = ops;

  return(v);
}

N_Vector N_VClone_Cuda(N_Vector w)
{
  N_Vector v;
  vector_type* wdat = static_cast<vector_type*>(w->content);
  vector_type* vdat = new vector_type(*wdat);
  v = NULL;
  v = N_VCloneEmpty_Cuda(w);
  if (v == NULL) return(NULL);

  v->content = vdat;

  return(v);
}


void N_VDestroy_Cuda(N_Vector v)
{
  vector_type* x = static_cast<vector_type*>(v->content);
  if (x != NULL) {
    delete x;
    v->content = NULL;
  }

  free(v->ops); v->ops = NULL;
  free(v); v = NULL;

  return;
}

void N_VSpace_Cuda(N_Vector X, sunindextype *lrw, sunindextype *liw)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  int npes;

  SUNMPI_Comm_size(comm, &npes);

  *lrw = getGlobalSize<realtype, sunindextype>(X);
  *liw = 2*npes;
}

void N_VConst_Cuda(realtype a, N_Vector X)
{
  vector_type *xvec = extract<realtype, sunindextype>(X);
  setConst(a, *xvec);
}

void N_VLinearSum_Cuda(realtype a, N_Vector X, realtype b, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *yvec = extract<realtype, sunindextype>(Y);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  linearSum(a, *xvec, b, *yvec, *zvec);
}

void N_VProd_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *yvec = extract<realtype, sunindextype>(Y);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  prod(*xvec, *yvec, *zvec);
}

void N_VDiv_Cuda(N_Vector X, N_Vector Y, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *yvec = extract<realtype, sunindextype>(Y);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  div(*xvec, *yvec, *zvec);
}

void N_VScale_Cuda(realtype a, N_Vector X, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  scale(a, *xvec, *zvec);
}

void N_VAbs_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  absVal(*xvec, *zvec);
}

void N_VInv_Cuda(N_Vector X, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  inv(*xvec, *zvec);
}

void N_VAddConst_Cuda(N_Vector X, realtype b, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  addConst(b, *xvec, *zvec);
}

realtype N_VDotProd_Cuda(N_Vector X, N_Vector Y)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *yvec = extract<realtype, sunindextype>(Y);

  realtype sum = dotProd(*xvec, *yvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return gsum;
}

realtype N_VMaxNorm_Cuda(N_Vector X)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);

  realtype locmax = maxNorm(*xvec);

  realtype globmax = SUNMPI_Allreduce_scalar(locmax, 2, comm);
  return globmax;
}

realtype N_VWrmsNorm_Cuda(N_Vector X, N_Vector W)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const sunindextype Nglob = getGlobalSize<realtype,sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *wvec = extract<realtype, sunindextype>(W);

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VWrmsNormMask_Cuda(N_Vector X, N_Vector W, N_Vector Id)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const sunindextype Nglob = getGlobalSize<realtype,sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *wvec = extract<realtype, sunindextype>(W);
  const vector_type *ivec = extract<realtype, sunindextype>(Id);

  realtype sum = wL2NormSquareMask(*xvec, *wvec, *ivec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return std::sqrt(gsum/Nglob);
}

realtype N_VMin_Cuda(N_Vector X)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);

  realtype locmin = findMin(*xvec);

  realtype globmin = SUNMPI_Allreduce_scalar(locmin, 3, comm);
  return globmin;
}

realtype N_VWL2Norm_Cuda(N_Vector X, N_Vector W)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  const vector_type *wvec = extract<realtype, sunindextype>(W);

  realtype sum = wL2NormSquare(*xvec, *wvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return std::sqrt(gsum);
}

realtype N_VL1Norm_Cuda(N_Vector X)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);

  realtype sum = L1Norm(*xvec);

  realtype gsum = SUNMPI_Allreduce_scalar(sum, 1, comm);
  return gsum;
}

void N_VCompare_Cuda(realtype c, N_Vector X, N_Vector Z)
{
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *zvec = extract<realtype, sunindextype>(Z);
  compare(c, *xvec, *zvec);
}

booleantype N_VInvTest_Cuda(N_Vector X, N_Vector Z)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *zvec = extract<realtype, sunindextype>(Z);

  realtype locmin = invTest(*xvec, *zvec);

  realtype globmin = SUNMPI_Allreduce_scalar(locmin, 3, comm);
  return (globmin < HALF);
}

/*
 * Creates mask for variables violating constraints
 */
booleantype N_VConstrMask_Cuda(N_Vector C, N_Vector X, N_Vector M)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(X);
  const vector_type *cvec = extract<realtype, sunindextype>(C);
  const vector_type *xvec = extract<realtype, sunindextype>(X);
  vector_type *mvec = extract<realtype, sunindextype>(M);

  realtype locsum = constrMask(*cvec, *xvec, *mvec);

  realtype globsum = SUNMPI_Allreduce_scalar(locsum, 1, comm);
  return (globsum < HALF);
}

realtype N_VMinQuotient_Cuda(N_Vector num, N_Vector denom)
{
  SUNMPI_Comm comm = getMPIComm<realtype, sunindextype>(num);
  const vector_type *numvec = extract<realtype, sunindextype>(num);
  const vector_type *denvec = extract<realtype, sunindextype>(denom);

  realtype locmin = minQuotient(*numvec, *denvec);

  realtype globmin = SUNMPI_Allreduce_scalar(locmin, 3, comm);
  return globmin;
}



} // extern "C"
